#include "hip/hip_runtime.h"
// Principle Component Background Subtraction 

#include "PCBS.h"
#include <iostream>

// Error check wrapper
inline 
hipError_t gpuErrchk(hipError_t result){
	if (result != hipSuccess){
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result==hipSuccess);
	}
	return result;
}

// inline void cuSolveErrchk(hipsolverStatus_t err, const char *file, const int line)
// {
// 	if (HIPSOLVER_STATUS_SUCCESS != err) {
// 		fprintf(stderr, "CUSOLVE error in file '%s', line %d, error: %s \nterminating!\n", __FILE__, __LINE__, \
// 			_cusolverGetErrorEnum(err)); \
// 			assert(0); \
// 	}
// }

// internal, potentially python side pre-processing
// However, given how often we will be running this, 
// it probably belongs here in a C loop. 
// TODO 
// void removeTemporalAvg()


/*
 *
 *
 * All references to LDA in the typical documentation are the same as rows here.
 */
extern "C"
void SVD(int width, 
  		 int height,
  		 int cols,					// e.g. number of frames
  		 float* modelFrames,
  		 float* targetFrame)
{
	int rows = width*height;
	float* hS;			// Host array for 
	float* dA; 			// Device ptr to Matrix A in the typical A = U x Sigma x VH ... SVD equation. 
	float* dU;
	float* dV;
	float* dS;
	int work;
	float* devWork;
	int *devInfo;

	// The flattened frames preceeding the target frame 
	gpuErrchk(hipMalloc(&dA, rows*cols*sizeof(float)));
	gpuErrchk(hipMemcpy(dA, modelFrames, rows*cols*sizeof(float), hipMemcpyHostToDevice));

	// The host space for storing the singular values, which we want to check against 
	 = (float*)malloc(std::min(rows, cols)*sizeof(float));

	// The device space for storing the Singular Vectors (columns of U and V)
	// as well as the Singular Values (S is just the diagonals of the Sigma matrix)

	gpuErrchk(hipMalloc(&dU, rows*rows*sizeof(float))); // Unitary numRows X numRows matrix
	gpuErrchk(hipMalloc(&dV, rows*cols*sizeof(float))); // 
	gpuErrchk(hipMalloc(&dS, std::min(rows, cols)*sizeof(float)));

	// Create solver instance
	hipsolverHandle_t solverHandle;
	hipsolverDnCreate(&solverHandle);

	// Get/allocate the amount of working space required for the algorithm through the API 
	hipsolverDnDgesvd_bufferSize(solverHandle, rows, cols, &work);
	gpuErrchk(hipMalloc(&devWork, work * sizeof(float)));
	gpuErrchk(hipDeviceSynchronize());

	// To check success. 
	gpuErrchk(hipMalloc(&devInfo, sizeof(int)));

	// Call it 
	auto error = hipsolverDnSgesvd(solverHandle,
				     'A', 'A',
				     rows,
				     cols,
				     dA,
				     rows,
				     dS,
				     dU,
				     rows,
				     dV,
				     cols,
				     devWork,
				     work,
				     NULL,
				     devInfo);

	gpuErrchk(hipDeviceSynchronize());
	std::cout << "happened" << std::endl;

	int hostInfo = 0;
	gpuErrchk(hipMemcpy(&hostInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	std::cout << "happened" << hostInfo << std::endl;

	if (hostInfo != 0) 
	{
		std::cout << "SVD device execution failed" << std::endl;
	}
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(hS, dS, std::min(rows, cols) * sizeof(float), hipMemcpyDeviceToHost));

	std::cout << "Singular values\n";
	for(int i = 0; i < min(rows, cols); i++)
	{
		std::cout << "dS["<<i<<"] = " << hS[i] << std::endl;
	}

	// Free stuff
	// TODO manage this python side, we need them later.
    if (dA      ) hipFree(dA);
    if (dS      ) hipFree(dS);
    if (dU      ) hipFree(dU);
    if (dV      ) hipFree(dV);
    if (devInfo ) hipFree(devInfo);
    if (devWork ) hipFree(devWork);
	hipsolverDnDestroy(solverHandle);
}

/* The dynamic version is meant to run with a "moving window". 
 * Simply, every call should add a frame to the model block 
 * for consideration in the eigen decomposition. If the model is */
// extern "C"
// void Dynamic_PCBS(int width, 
// 		  		 int height,
// 		  		 int depth,
// 		  		 float* modelFrames,
// 		  		 float* targetFrame,
// 		  		 )
// {
// 	// Create 
// 	hipsolverHandle_t cusolverH;
// 	gpuErrchk(hipsolverDnCreate(&solverHandle));

// 	gpuErrchk(hipsolverDnDestroy(&solverHandle));
// }