#include "hip/hip_runtime.h"
// Principle Component Background Subtraction 

#include "PCBS.h"
#include <iostream>

// Error check wrapper
inline 
hipError_t gpuErrchk(hipError_t result){
	if (result != hipSuccess){
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result==hipSuccess);
	}
	return result;
}

// inline void cuSolveErrchk(hipsolverStatus_t err, const char *file, const int line)
// {
// 	if (HIPSOLVER_STATUS_SUCCESS != err) {
// 		fprintf(stderr, "CUSOLVE error in file '%s', line %d, error: %s \nterminating!\n", __FILE__, __LINE__, \
// 			_cusolverGetErrorEnum(err)); \
// 			assert(0); \
// 	}
// }

// internal, potentially python side pre-processing
// However, given how often we will be running this, 
// it probably belongs here in a C loop. 
// TODO 
// void removeTemporalAvg()


/*
 *
 *
 */
extern "C"
void Static_PCBS(int width, 
		  		 int height,
		  		 int cols,					// e.g. number of frames
		  		 float* modelFrames,
		  		 float* targetFrame)
{
	// The flattened frames preceeding the target frame comprise the Matrix A 
	// in the typical A = U x Sigma x VH ... SVD equation. 
	float* dA; 
	int rows = width*height;
	gpuErrchk(hipMalloc(&dA, rows*cols*sizeof(float)));
	gpuErrchk(hipMemcpy(&dA, modelFrames, rows*cols*sizeof(float), hipMemcpyHostToDevice));

	// The host space for storing the singular values, which we want to check against 
	float *hS;

	// The device space for storing the Singular Vectors (columns of U and V)
	// as well as the Singular Values (S is just the diagonals of the Sigma matrix)
	float *dU;
	float *dV;
	float *dS;
	gpuErrchk(hipMalloc(&dU, rows*sizeof(float))); // Unitary numRows X numRows matrix
	gpuErrchk(hipMalloc(&dV, cols*sizeof(float))); // 
	gpuErrchk(hipMalloc(&dS, std::min(rows, cols)*sizeof(float)));

	// Create solver instance
	hipsolverHandle_t solverHandle;
	hipsolverDnCreate(&solverHandle);

	// Get/allocate the amount of working space required for the algorithm through the API 
	int work;
	hipsolverDnDgesvd_bufferSize(solverHandle, rows, cols, &work);
	float* devWork;
	gpuErrchk(hipMalloc(&devWork, work * sizeof(float)));

	// To check success. 
	int *devInfo;
	gpuErrchk(hipMalloc(&devInfo, sizeof(int)));

	// Call it 
	hipsolverDnSgesvd(solverHandle,
				     'A', 'A',
				     rows,
				     cols,
				     dA,
				     rows,
				     dS,
				     dU,
				     rows,
				     dV,
				     cols,
				     devWork,
				     work,
				     NULL,
				     devInfo);

	int hostInfo = 0;
	gpuErrchk(hipMemcpy(&hostInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	if (hostInfo != 0) 
	{
		std::cout << "SVD device execution failed" << std::endl;
	}
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(hS, dS, std::min(rows, cols) * sizeof(float), hipMemcpyDeviceToHost));

	std::cout << "Singular values\n";
	for(int i = 0; i < min(rows, cols); i++)
	{
		std::cout << "dS["<<i<<"] = " << hS[i] << std::endl;
	}

	// Free stuff
    if (dA      ) hipFree(dA);
    if (dS      ) hipFree(dS);
    if (dU      ) hipFree(dU);
    if (dV      ) hipFree(dV);
    if (devInfo ) hipFree(devInfo);
    if (devWork ) hipFree(devWork);
	hipsolverDnDestroy(solverHandle);
}

/* The dynamic version is meant to run with a "moving window". 
 * Simply, every call should add a frame to the model block 
 * for consideration in the eigen decomposition. If the model is */
// extern "C"
// void Dynamic_PCBS(int width, 
// 		  		 int height,
// 		  		 int depth,
// 		  		 float* modelFrames,
// 		  		 float* targetFrame,
// 		  		 )
// {
// 	// Create 
// 	hipsolverHandle_t cusolverH;
// 	gpuErrchk(hipsolverDnCreate(&solverHandle));

// 	gpuErrchk(hipsolverDnDestroy(&solverHandle));
// }